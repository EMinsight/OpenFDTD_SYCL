#include "hip/hip_runtime.h"
/*
murH_gpu.cu
*/

#include "ofd.h"
#include "ofd_cuda.h"


__host__ __device__
static void murh(real_t *h, mur_t *q, int64_t ni, int64_t nj, int64_t nk, int64_t n0)
{
	const int64_t m0 = (ni * q->i)  + (nj * q->j)  + (nk * q->k)  + n0;
	const int64_t m1 = (ni * q->i1) + (nj * q->j1) + (nk * q->k1) + n0;

	h[m0] = q->f + q->g * (h[m1] - h[m0]);
	q->f = h[m1];
}


__global__
static void murh_gpu(
	int64_t num, real_t *h, mur_t *fmur,
	int64_t ni, int64_t nj, int64_t nk, int64_t n0)
{
	const int64_t n = threadIdx.x + (blockIdx.x * blockDim.x);
	if (n < num) {
		murh(h, &fmur[n], ni, nj, nk, n0);
	}
}


static void murh_cpu(
	int64_t num, real_t *h, mur_t *fmur,
	int64_t ni, int64_t nj, int64_t nk, int64_t n0)
{
	for (int64_t n = 0; n < num; n++) {
		murh(h, &fmur[n], ni, nj, nk, n0);
	}
}


void murH(int64_t num, mur_t *fmur, real_t *h)
{
	if (num <= 0) return;

	if (GPU) {
		murh_gpu<<<(int)CEIL(num, murBlock), murBlock>>>(
			num, h, fmur,
			Ni, Nj, Nk, N0);
		if (UM) hipDeviceSynchronize();
	}
	else {
		murh_cpu(
			num, h, fmur,
			Ni, Nj, Nk, N0);
	}
}
