#include "hip/hip_runtime.h"
/*
updateEz.cu

update Ez
*/

#include "ofd.h"
#include "ofd_cuda.h"
#include "finc_cuda.h"


__host__ __device__
static void updateEz_f_vector(
	int i, int j, int k,
	real_t ez[], const real_t hx[], const real_t hy[],
	const real_t k1[], const real_t k2[],
	real_t rxn, real_t ryn, param_t *p)
{
	const int64_t n = LA(p, i, j, k);

	ez[n] = k1[n] * ez[n]
	      + k2[n] * (rxn * (hy[n] - hy[n - p->Ni])
	               - ryn * (hx[n] - hx[n - p->Nj]));
}


__host__ __device__
static void updateEz_f_no_vector(
	int i, int j, int k,
	real_t ez[], const real_t hx[], const real_t hy[], const id_t iez[],
	const real_t c1[], const real_t c2[],
	real_t rxn, real_t ryn, param_t *p)
{
	const int64_t n = LA(p, i, j, k);
	const int64_t m = iez[n];

	ez[n] = c1[m] * ez[n]
	      + c2[m] * (rxn * (hy[n] - hy[n - p->Ni])
	               - ryn * (hx[n] - hx[n - p->Nj]));
}


__host__ __device__
static void updateEz_p_vector(
	int i, int j, int k,
	real_t ez[], const real_t hx[], const real_t hy[], const id_t iez[],
	const real_t k1[], const real_t k2[],
	real_t rxn, real_t ryn, param_t *p,
	real_t x, real_t y, real_t z, real_t t)
{
	const int64_t n = LA(p, i, j, k);
	const int64_t m = iez[n];

	if (m == 0) {
		ez[n] += rxn * (hy[n] - hy[n - p->Ni])
		       - ryn * (hx[n] - hx[n - p->Nj]);
	}
	else {
		real_t fi, dfi;
		finc_cuda(x, y, z, t, p->r0, p->ri, p->ei[2], p->ai, p->dt, &fi, &dfi);
		if (m == PEC) {
			ez[n] = -fi;
		}
		else {
			ez[n] = k1[n] * ez[n]
			      + k2[n] * (rxn * (hy[n] - hy[n - p->Ni])
			               - ryn * (hx[n] - hx[n - p->Nj]))
			      - (k1[n] - k2[n]) * dfi
			      - (1 - k1[n]) * fi;
		}
	}
}


__host__ __device__
static void updateEz_p_no_vector(
	int i, int j, int k,
	real_t ez[], const real_t hx[], const real_t hy[], const id_t iez[],
	const real_t c1[], const real_t c2[],
	real_t rxn, real_t ryn, param_t *p,
	real_t x, real_t y, real_t z, real_t t)
{
	const int64_t n = LA(p, i, j, k);
	const int64_t m = iez[n];

	if (m == 0) {
		ez[n] += rxn * (hy[n] - hy[n - p->Ni])
		       - ryn * (hx[n] - hx[n - p->Nj]);
	}
	else {
		real_t fi, dfi;
		finc_cuda(x, y, z, t, p->r0, p->ri, p->ei[2], p->ai, p->dt, &fi, &dfi);
		if (m == PEC) {
			ez[n] = -fi;
		}
		else {
			ez[n] = c1[m] * ez[n]
			      + c2[m] * (rxn * (hy[n] - hy[n - p->Ni])
			               - ryn * (hx[n] - hx[n - p->Nj]))
			      - (c1[m] - c2[m]) * dfi
			      - (1 - c1[m]) * fi;
		}
	}
}


__global__
static void updateEz_gpu(int vector,
	real_t ez[], const real_t hx[], const real_t hy[], const id_t iez[],
	const real_t c1[], const real_t c2[], const real_t k1[], const real_t k2[],
	const real_t rxn[], const real_t ryn[], const real_t xn[], const real_t yn[], const real_t zc[], real_t t)
{
	const int i = d_Param.iMin + threadIdx.z + (blockIdx.z * blockDim.z);
	const int j = d_Param.jMin + threadIdx.y + (blockIdx.y * blockDim.y);
	const int k = d_Param.kMin + threadIdx.x + (blockIdx.x * blockDim.x);
	if ((i <= d_Param.iMax) &&
	    (j <= d_Param.jMax) &&
	    (k <  d_Param.kMax)) {
		if (d_Param.NFeed) {
			if (vector) {
				updateEz_f_vector(
					i, j, k,
					ez, hx, hy,
					k1, k2,
					rxn[i], ryn[j], &d_Param);
			}
			else {
				updateEz_f_no_vector(
					i, j, k,
					ez, hx, hy, iez,
					c1, c2,
					rxn[i], ryn[j], &d_Param);
			}
		}
		else if (d_Param.IPlanewave) {
			if (vector) {
				updateEz_p_vector(
					i, j, k,
					ez, hx, hy, iez,
					k1, k2,
					rxn[i], ryn[j], &d_Param,
					xn[i], yn[j], zc[k], t);
			}
			else {
				updateEz_p_no_vector(
					i, j, k,
					ez, hx, hy, iez,
					c1, c2,
					rxn[i], ryn[j], &d_Param,
					xn[i], yn[j], zc[k], t);
			}
		}
	}
}


static void updateEz_cpu(
	real_t ez[], const real_t hx[], const real_t hy[], const id_t iez[],
	const real_t c1[], const real_t c2[],
	const real_t rxn[], const real_t ryn[], const real_t xn[], const real_t yn[], const real_t zc[], real_t t)
{
	for (int i = h_Param.iMin; i <= h_Param.iMax; i++) {
	for (int j = h_Param.jMin; j <= h_Param.jMax; j++) {
	for (int k = h_Param.kMin; k <  h_Param.kMax; k++) {
		if (h_Param.NFeed) {
			updateEz_f_no_vector(
				i, j, k,
				ez, hx, hy, iez,
				c1, c2,
				rxn[i], ryn[j], &h_Param);
		}
		else if (h_Param.IPlanewave) {
			updateEz_p_no_vector(
				i, j, k,
				ez, hx, hy, iez,
				c1, c2,
				rxn[i], ryn[j], &h_Param,
				xn[i], yn[j], zc[k], t);
		}
	}
	}
	}
}


void updateEz(double t)
{
	if (GPU) {
		hipMemcpyToSymbol(HIP_SYMBOL(d_Param), &h_Param, sizeof(param_t));
		dim3 grid(
			CEIL(kMax - kMin + 0, updateBlock.x),
			CEIL(jMax - jMin + 1, updateBlock.y),
			CEIL(iMax - iMin + 1, updateBlock.z));
		updateEz_gpu<<<grid, updateBlock>>>(VECTOR,
			Ez, Hx, Hy, d_iEz,
			d_C1, d_C2, d_K1Ez, d_K2Ez,
			d_RXn, d_RYn, d_Xn, d_Yn, d_Zc, (real_t)t);
		if (UM) hipDeviceSynchronize();
	}
	else {
		// CPU : no-vector only
		updateEz_cpu(
			Ez, Hx, Hy, iEz,
			C1, C2,
			RXn, RYn, h_Xn, h_Yn, h_Zc, (real_t)t);
	}
}
