#include "hip/hip_runtime.h"
/*
updateHx.cu

update Hx
*/

#include "ofd.h"
#include "ofd_cuda.h"
#include "finc_cuda.h"


__host__ __device__
static void updateHx_f_vector(
	int i, int j, int k,
	real_t hx[], const real_t ey[], const real_t ez[],
	const real_t k1[], const real_t k2[],
	real_t ryc, real_t rzc, param_t *p)
{
	const int64_t n = LA(p, i, j, k);

	hx[n] = k1[n] * hx[n]
	      - k2[n] * (ryc * (ez[n + p->Nj] - ez[n])
	               - rzc * (ey[n + p->Nk] - ey[n]));
}


__host__ __device__
static void updateHx_f_no_vector(
	int i, int j, int k,
	real_t hx[], const real_t ey[], const real_t ez[], const id_t ihx[],
	const real_t d1[], const real_t d2[],
	real_t ryc, real_t rzc, param_t *p)
{
	const int64_t n = LA(p, i, j, k);
	const int64_t m = ihx[n];

	hx[n] = d1[m] * hx[n]
	      - d2[m] * (ryc * (ez[n + p->Nj] - ez[n])
	               - rzc * (ey[n + p->Nk] - ey[n]));
}


__host__ __device__
static void updateHx_p_vector(
	int i, int j, int k,
	real_t hx[], const real_t ey[], const real_t ez[], const id_t ihx[],
	const real_t k1[], const real_t k2[],
	real_t ryc, real_t rzc, param_t *p,
	real_t x, real_t y, real_t z, real_t t)
{
	const int64_t n = LA(p, i, j, k);
	const int64_t m = ihx[n];

	if (m == 0) {
		hx[n] -= ryc * (ez[n + p->Nj] - ez[n])
		       - rzc * (ey[n + p->Nk] - ey[n]);
	}
	else {
		real_t fi, dfi;
		finc_cuda(x, y, z, t, p->r0, p->ri, p->hi[0], p->ai, p->dt, &fi, &dfi);
		if (m == PEC) {
			hx[n] = -fi;
		}
		else {
			hx[n] = k1[n] * hx[n]
			      - k2[n] * (ryc * (ez[n + p->Nj] - ez[n])
			               - rzc * (ey[n + p->Nk] - ey[n]))
			      - (k1[n] - k2[n]) * dfi
			      - (1 - k1[n]) * fi;
		}
	}
}


__host__ __device__
static void updateHx_p_no_vector(
	int i, int j, int k,
	real_t hx[], const real_t ey[], const real_t ez[], const id_t ihx[],
	const real_t d1[], const real_t d2[],
	real_t ryc, real_t rzc, param_t *p,
	real_t x, real_t y, real_t z, real_t t)
{
	const int64_t n = LA(p, i, j, k);
	const int64_t m = ihx[n];

	if (m == 0) {
		hx[n] -= ryc * (ez[n + p->Nj] - ez[n])
		       - rzc * (ey[n + p->Nk] - ey[n]);
	}
	else {
		real_t fi, dfi;
		finc_cuda(x, y, z, t, p->r0, p->ri, p->hi[0], p->ai, p->dt, &fi, &dfi);
		if (m == PEC) {
			hx[n] = -fi;
		}
		else {
			hx[n] = d1[m] * hx[n]
			      - d2[m] * (ryc * (ez[n + p->Nj] - ez[n])
			               - rzc * (ey[n + p->Nk] - ey[n]))
			      - (d1[m] - d2[m]) * dfi
			      - (1 - d1[m]) * fi;
		}
	}
}


__global__
static void updateHx_gpu(int vector,
	real_t hx[], const real_t ey[], const real_t ez[], const id_t ihx[],
	const real_t d1[], const real_t d2[], const real_t k1[], const real_t k2[],
	const real_t ryc[], const real_t rzc[], const real_t xn[], const real_t yc[], const real_t zc[], real_t t)
{
	const int i = d_Param.iMin + threadIdx.z + (blockIdx.z * blockDim.z);
	const int j = d_Param.jMin + threadIdx.y + (blockIdx.y * blockDim.y);
	const int k = d_Param.kMin + threadIdx.x + (blockIdx.x * blockDim.x);
	if ((i <= d_Param.iMax) &&
	    (j <  d_Param.jMax) &&
	    (k <  d_Param.kMax)) {
		if (d_Param.NFeed) {
			if (vector) {
				updateHx_f_vector(
					i, j, k,
					hx, ey, ez,
					k1, k2,
					ryc[j], rzc[k], &d_Param);
			}
			else {
				updateHx_f_no_vector(
					i, j, k,
					hx, ey, ez, ihx,
					d1, d2,
					ryc[j], rzc[k], &d_Param);
			}
		}
		else if (d_Param.IPlanewave) {
			if (vector) {
				updateHx_p_vector(
					i, j, k,
					hx, ey, ez, ihx,
					k1, k2,
					ryc[j], rzc[k], &d_Param,
					xn[i], yc[j], zc[k], t);
			}
			else {
				updateHx_p_no_vector(
					i, j, k,
					hx, ey, ez, ihx,
					d1, d2,
					ryc[j], rzc[k], &d_Param,
					xn[i], yc[j], zc[k], t);
			}
		}
	}
}


static void updateHx_cpu(
	real_t hx[], const real_t ey[], const real_t ez[], const id_t ihx[],
	const real_t d1[], const real_t d2[],
	const real_t ryc[], const real_t rzc[], const real_t xn[], const real_t yc[], const real_t zc[], real_t t)
{
	for (int i = h_Param.iMin; i <= h_Param.iMax; i++) {
	for (int j = h_Param.jMin; j <  h_Param.jMax; j++) {
	for (int k = h_Param.kMin; k <  h_Param.kMax; k++) {
		if (h_Param.NFeed) {
			updateHx_f_no_vector(
				i, j, k,
				hx, ey, ez, ihx,
				d1, d2,
				ryc[j], rzc[k], &h_Param);
		}
		else if (h_Param.IPlanewave) {
			updateHx_p_no_vector(
				i, j, k,
				hx, ey, ez, ihx,
				d1, d2,
				ryc[j], rzc[k], &h_Param,
				xn[i], yc[j], zc[k], t);
		}
	}
	}
	}
}


void updateHx(double t)
{
	if (GPU) {
		hipMemcpyToSymbol(HIP_SYMBOL(d_Param), &h_Param, sizeof(param_t));
		dim3 grid(
			CEIL(kMax - kMin + 0, updateBlock.x),
			CEIL(jMax - jMin + 0, updateBlock.y),
			CEIL(iMax - iMin + 1, updateBlock.z));
		updateHx_gpu<<<grid, updateBlock>>>(VECTOR,
			Hx, Ey, Ez, d_iHx,
			d_D1, d_D2, d_K1Hx, d_K2Hx,
			d_RYc, d_RZc, d_Xn, d_Yc, d_Zc, (real_t)t);
		if (UM) hipDeviceSynchronize();
	}
	else {
		// CPU : no-vector only
		updateHx_cpu(
			Hx, Ey, Ez, iHx,
			D1, D2,
			RYc, RZc, h_Xn, h_Yc, h_Zc, (real_t)t);
	}
}
